#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void add_kernel(const float *a, const float *b, float *c, size_t N) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

torch::Tensor add_cuda(torch::Tensor a, torch::Tensor b) {
    auto c = torch::empty_like(a);
    const auto N = a.numel();
    const int threads = 1024;
    const int blocks = (N + threads - 1) / threads;

    AT_ASSERTM(a.is_cuda(), "Tensor a must be a CUDA tensor");
    AT_ASSERTM(b.is_cuda(), "Tensor b must be a CUDA tensor");
    add_kernel<<<blocks, threads>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), N);

    return c;
}

__global__ void computeValueKernel(float* temp_sum, float* current_image_estimate, float* kappa, int width, int height, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return; // Boundary check

    int inputIndex = z * width * height + y * width + x;

    float sum = 0.0f;
    float diff = 0.0f;
    float add = 0.0f;

    // Apply convolution kernel hard coded 3x3x3 neighbourhood with unity weights
    for(int dz = -1; dz <= 1; dz++) {
        for(int dy = -1; dy <= 1; dy++) {
            for(int dx = -1; dx <= 1; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;
                // Clamp to image boundary, i.e. replicate boundary
                nx = min(max(nx, 0), width - 1); 
                ny = min(max(ny, 0), height - 1);
                nz = min(max(nz, 0), depth - 1);

                int neighbourIndex = nz * width * height + ny * width + nx;

                diff = (current_image_estimate[inputIndex] - current_image_estimate[neighbourIndex]);
                add = (current_image_estimate[inputIndex] + current_image_estimate[neighbourIndex]);
                sum -= pow(diff, 2)/(add + 2*abs(diff) + 1e-9);
                
            }
        }
    }
    // Use atomicAdd to safely accumulate the sum into a global variable
    temp_sum[inputIndex] = sum; // * kappa[inputIndex];
}

torch::Tensor computeValueCuda(torch::Tensor current_image_estimate, torch::Tensor kappa) {
    AT_ASSERTM(current_image_estimate.is_cuda(), "Tensor current_image_estimate must be a CUDA tensor");
    AT_ASSERTM(kappa.is_cuda(), "Tensor kappa must be a CUDA tensor");
    int width = current_image_estimate.size(2);
    int height = current_image_estimate.size(1);
    int depth = current_image_estimate.size(0);

    // Create a temporary array to store the sum of the values
    auto options = current_image_estimate.options();
    auto temp_sum = torch::zeros({depth, height, width}, options);
    
    // Launch the kernel
    dim3 threads = dim3(9, 9, 9);
    dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, (depth + threads.z - 1) / threads.z);
    computeValueKernel<<<blocks, threads>>>(temp_sum.data_ptr<float>(), current_image_estimate.data_ptr<float>(), kappa.data_ptr<float>(), width, height, depth);
    
    // Ensure the kernel execution completes
    hipDeviceSynchronize();

    // Sum the temporary array values
    auto value = torch::sum(temp_sum);

    return value;
}

__global__ void computeGradientKernel(float* gradient, float* current_image_estimate, float* kappa, int width, int height, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return; // Boundary check

    int inputIndex = z * width * height + y * width + x;

    float voxel_gradient = 0.0f;
    float diff = 0.0f;
    float diff_abs = 0.0f;
    float add_3 = 0.0f;
    float add = 0.0f;

    // Apply convolution kernel hard coded 3x3x3 neighbourhood with unity weights
    for(int dz = -1; dz <= 1; dz++) {
        for(int dy = -1; dy <= 1; dy++) {
            for(int dx = -1; dx <= 1; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;
                // Clamp to image boundary, i.e. replicate boundary
                nx = min(max(nx, 0), width - 1); 
                ny = min(max(ny, 0), height - 1);
                nz = min(max(nz, 0), depth - 1);

                int neighbourIndex = nz * width * height + ny * width + nx;

                diff = (current_image_estimate[inputIndex] - current_image_estimate[neighbourIndex]);
                diff_abs = abs(diff);
                add = (current_image_estimate[inputIndex] + current_image_estimate[neighbourIndex]);
                add_3 = (current_image_estimate[inputIndex] + 3*current_image_estimate[neighbourIndex]);
                voxel_gradient -= (diff*(2*diff_abs + add_3))/pow(add + 2*diff_abs + 1e-9, 2);
                
            }
        }
    }
    // Use atomicAdd to safely accumulate the sum into a global variable
    gradient[inputIndex] = voxel_gradient; // * kappa[inputIndex];
}

torch::Tensor computeGradientCuda(torch::Tensor current_image_estimate, torch::Tensor kappa) {
    AT_ASSERTM(current_image_estimate.is_cuda(), "Tensor current_image_estimate must be a CUDA tensor");
    AT_ASSERTM(kappa.is_cuda(), "Tensor kappa must be a CUDA tensor");
    int width = current_image_estimate.size(2);
    int height = current_image_estimate.size(1);
    int depth = current_image_estimate.size(0);

    // Create a temporary array to store the sum of the values
    auto options = current_image_estimate.options();
    auto gradient = torch::zeros({depth, height, width}, options);
    
    // Launch the kernel
    dim3 threads = dim3(9, 9, 9);
    dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, (depth + threads.z - 1) / threads.z);
    computeGradientKernel<<<blocks, threads>>>(gradient.data_ptr<float>(), current_image_estimate.data_ptr<float>(), kappa.data_ptr<float>(), width, height, depth);
    
    // Ensure the kernel execution completes
    hipDeviceSynchronize();

    return gradient;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("add", &add_cuda, "Element-wise addition of two tensors");
    m.def("compute_value", &computeValueCuda, "Compute RDP value");
    m.def("compute_gradient", &computeGradientCuda, "Compute RDP gradient");
}