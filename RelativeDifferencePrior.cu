#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void add_kernel(const float *a, const float *b, float *c, size_t N) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

torch::Tensor add_cuda(torch::Tensor a, torch::Tensor b) {
    auto c = torch::empty_like(a);
    const auto N = a.numel();
    const int threads = 1024;
    const int blocks = (N + threads - 1) / threads;

    AT_ASSERTM(a.is_cuda(), "Tensor a must be a CUDA tensor");
    AT_ASSERTM(b.is_cuda(), "Tensor b must be a CUDA tensor");
    add_kernel<<<blocks, threads>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), N);

    return c;
}

__global__ void computeValueKernel(float* value, float* current_image_estimate, float* kappa, int width, int height, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || z >= depth) return; // Boundary check

    float sum = 0.0f;
    float diff = 0.0f;
    float add = 0.0f;

    // Apply convolution kernel hard coded 3x3x3 neighbourhood with unity weights
    for(int dz = -1; dz <= 1; dz++) {
        for(int dy = -1; dy <= 1; dy++) {
            for(int dx = -1; dx <= 1; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                int nz = z + dz;

                // Boundary check for the volume
                if(nx > 0 && nx < width && ny > 0 && ny < height && nz > 0 && nz < depth) {
                    int inputIndex = nz * width * height + ny * width + nx;
                    int neighbourIndex = z * width * height +  y * width + x;
                    diff = (current_image_estimate[inputIndex] - current_image_estimate[neighbourIndex]);
                    add = (current_image_estimate[inputIndex] + current_image_estimate[neighbourIndex]);
                    sum -= pow(diff, 2)/(add + 2*abs(diff) + 1e-9);
                }
            }
        }
    }
    // Use atomicAdd to safely accumulate the sum into a global variable
    atomicAdd(value, sum);
}

torch::Tensor computeValueCuda(torch::Tensor current_image_estimate, torch::Tensor kappa) {
    AT_ASSERTM(current_image_estimate.is_cuda(), "Tensor current_image_estimate must be a CUDA tensor");
    AT_ASSERTM(kappa.is_cuda(), "Tensor kappa must be a CUDA tensor");

    int width = current_image_estimate.size(2);
    int height = current_image_estimate.size(1);
    int depth = current_image_estimate.size(0);
    // print input options
    std::cout << "Input options: " << current_image_estimate.options() << std::endl;
    auto value = torch::zeros({1}, current_image_estimate.options());
    dim3 threads = dim3(9, 9, 9);
    dim3 blocks = dim3((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, (depth + threads.z - 1) / threads.z);
    computeValueKernel<<<blocks, threads>>>(value.data_ptr<float>(), current_image_estimate.data_ptr<float>(), kappa.data_ptr<float>(), width, height, depth);
    return value;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("add", &add_cuda, "Element-wise addition of two tensors");
    m.def("compute_value", &computeValueCuda, "Compute RDP value");
}