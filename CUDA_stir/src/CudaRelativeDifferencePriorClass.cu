#include <vector>
#include "CudaKernels.h"

// Wrapper for computeRelativeDifferencePriorGradientKernel
void runGradientKernelOnCPUVectors(const std::vector<float>& tmp_grad, const std::vector<float>& image, const std::vector<float>& weights, const std::vector<float>& kappa, const std::vector<float>& penalisation_factor, const std::vector<float>& gamma, const std::vector<float>& epsilon, const int z_dim, const int y_dim, const int x_dim) {
    // Allocate device memory
    float* d_tmp_grad;
    float* d_image;
    float* d_weights;
    float* d_kappa;
    float* d_penalisation_factor;
    float* d_gamma;
    float* d_epsilon;
    size_t size = tmp_grad.size() * sizeof(float);
    hipMalloc((void**)&d_tmp_grad, size);
    hipMalloc((void**)&d_image, size);
    hipMalloc((void**)&d_weights, size);
    hipMalloc((void**)&d_kappa, size);
    hipMalloc((void**)&d_penalisation_factor, size);
    hipMalloc((void**)&d_gamma, size);
    hipMalloc((void**)&d_epsilon, size);

    // Copy vectors from host to device memory
    hipMemcpy(d_tmp_grad, tmp_grad.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_image, image.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_kappa, kappa.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_penalisation_factor, penalisation_factor.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_gamma, gamma.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_epsilon, epsilon.data(), size, hipMemcpyHostToDevice);

    // Calculate the number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (tmp_grad.size() + threadsPerBlock - 1) / threadsPerBlock;

    // Run the kernel function
    computeRelativeDifferencePriorGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tmp_grad, d_image, d_weights, d_kappa, d_penalisation_factor, d_gamma, d_epsilon, z_dim, y_dim, x_dim);

    // Free device memory
    hipFree(d_tmp_grad);
    hipFree(d_image);
    hipFree(d_weights);
    hipFree(d_kappa);
    hipFree(d_penalisation_factor);
    hipFree(d_gamma);
    hipFree(d_epsilon);
}

// Wrapper for computeRelativeDifferencePriorValueKernel
void runValueKernelOnCPUVectors(const std::vector<float>& tmp_value, const std::vector<float>& image, const std::vector<float>& weights, const std::vector<float>& kappa, const std::vector<float>& penalisation_factor, const std::vector<float>& gamma, const std::vector<float>& epsilon, const int z_dim, const int y_dim, const int x_dim) {
    // Similar to the above function, allocate device memory, copy data, run the kernel, and free memory.
    // Allocate device memory
    float* d_tmp_grad;
    float* d_image;
    float* d_weights;
    float* d_kappa;
    float* d_penalisation_factor;
    float* d_gamma;
    float* d_epsilon;
    size_t size = tmp_value.size() * sizeof(float);
    hipMalloc((void**)&d_tmp_grad, size);
    hipMalloc((void**)&d_image, size);
    hipMalloc((void**)&d_weights, size);
    hipMalloc((void**)&d_kappa, size);
    hipMalloc((void**)&d_penalisation_factor, size);
    hipMalloc((void**)&d_gamma, size);
    hipMalloc((void**)&d_epsilon, size);

    // Copy vectors from host to device memory
    hipMemcpy(d_tmp_grad, tmp_value.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_image, image.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_kappa, kappa.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_penalisation_factor, penalisation_factor.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_gamma, gamma.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_epsilon, epsilon.data(), size, hipMemcpyHostToDevice);

    // Calculate the number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (tmp_value.size() + threadsPerBlock - 1) / threadsPerBlock;

    // Run the kernel function
    computeRelativeDifferencePriorValueKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tmp_grad, d_image, d_weights, d_kappa, d_penalisation_factor, d_gamma, d_epsilon, z_dim, y_dim, x_dim);

    // Free device memory
    hipFree(d_tmp_grad);
    hipFree(d_image);
    hipFree(d_weights);
    hipFree(d_kappa);
    hipFree(d_penalisation_factor);
    hipFree(d_gamma);
    hipFree(d_epsilon);
}